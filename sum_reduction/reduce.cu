#include "hip/hip_runtime.h"
#include "reduce.h"

#include <cmath>

#include "hip/hip_runtime.h"
#include ""
#include "utils.h"

__global__
void block_sum_reduce(unsigned int* const d_block_sums, 
	const unsigned int* const d_in,
	const unsigned int d_in_len)
{
	extern __shared__ unsigned int s_out[];

	unsigned int glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	// Zero out shared memory
	// Especially important when padding shmem for
	//  non-power of 2 sized input
	s_out[2 * threadIdx.x] = 0;
	s_out[2 * threadIdx.x + 1] = 0;

	__syncthreads();

	// Copy d_in to shared memory per block
	if (2 * glbl_t_idx < d_in_len)
	{
		s_out[2 * threadIdx.x] = d_in[2 * glbl_t_idx];
		if (2 * glbl_t_idx + 1 < d_in_len)
			s_out[2 * threadIdx.x + 1] = d_in[2 * glbl_t_idx + 1];
	}

	__syncthreads();

	// 2^11 = 2048, the max amount of data a block can blelloch scan
	unsigned int max_steps = 11;

	unsigned int r_idx = 0;
	unsigned int l_idx = 0;
	unsigned int sum = 0; // global sum can be passed to host if needed
	unsigned int t_active = 0;
	for (unsigned int s = 0; s < max_steps; ++s)
	{
		t_active = 0;

		// calculate necessary indexes
		// right index must be (t+1) * 2^(s+1)) - 1
		r_idx = ((threadIdx.x + 1) * (1 << (s + 1))) - 1;
		if (r_idx >= 0 && r_idx < 2048)
			t_active = 1;

		if (t_active)
		{
			// left index must be r_idx - 2^s
			l_idx = r_idx - (1 << s);

			// do the actual add operation
			sum = s_out[l_idx] + s_out[r_idx];
		}
		__syncthreads();

		if (t_active)
			s_out[r_idx] = sum;
		__syncthreads();
	}

	// Copy last element (total sum of block) to block sums array
	// Then, reset last element to operation's identity (sum, 0)
	if (threadIdx.x == 0)
	{
		d_block_sums[blockIdx.x] = s_out[r_idx];
	}
}

__global__ void reduce0(unsigned int* g_odata, unsigned int* g_idata, unsigned int len) {
	extern __shared__ unsigned int sdata[];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = 0;

	if (i < len)
	{
		sdata[tid] = g_idata[i];
	}

	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) 
		g_odata[blockIdx.x] = sdata[0];
}

void print_d_array(unsigned int* d_array, unsigned int len)
{
	unsigned int* h_array = new unsigned int[len];
	checkCudaErrors(hipMemcpy(h_array, d_array, sizeof(unsigned int) * len, hipMemcpyDeviceToHost));
	for (unsigned int i = 0; i < len; ++i)
	{
		std::cout << h_array[i] << " ";
	}
	std::cout << std::endl;

	delete[] h_array;
}

unsigned int gpu_sum_reduce(unsigned int* d_in, unsigned int d_in_len)
{
	unsigned int total_sum = 0;

	// Set up number of threads and blocks
	// If input size is not power of two, the remainder will still need a whole block
	// Thus, number of blocks must be the least number of 2048-blocks greater than the input size
	unsigned int block_sz = MAX_BLOCK_SZ;
	// our block_sum_reduce()
	//unsigned int max_elems_per_block = block_sz * 2; // due to binary tree nature of algorithm
	// NVIDIA's reduceX()
	unsigned int max_elems_per_block = block_sz;
	
	unsigned int grid_sz = 0;
	if (d_in_len <= max_elems_per_block)
	{
		grid_sz = (unsigned int)std::ceil(float(d_in_len) / float(max_elems_per_block));
	}
	else
	{
		grid_sz = d_in_len / max_elems_per_block;
		if (d_in_len % max_elems_per_block != 0)
			grid_sz++;
	}

	// Allocate memory for array of total sums produced by each block
	// Array length must be the same as number of blocks / grid size
	unsigned int* d_block_sums;
	checkCudaErrors(hipMalloc(&d_block_sums, sizeof(unsigned int) * grid_sz));
	checkCudaErrors(hipMemset(d_block_sums, 0, sizeof(unsigned int) * grid_sz));

	// Sum data allocated for each block
	//block_sum_reduce<<<grid_sz, block_sz, sizeof(unsigned int) * max_elems_per_block>>>(d_block_sums, d_in, d_in_len);
	reduce0<<<grid_sz, block_sz, sizeof(unsigned int) * block_sz>>>(d_block_sums, d_in, d_in_len);
	//print_d_array(d_block_sums, grid_sz);

	// Sum each block's total sums (to get global total sum)
	// Use basic implementation if number of total sums is <= 2048
	// Else, recurse on this same function
	if (grid_sz <= max_elems_per_block)
	{
		unsigned int* d_total_sum;
		checkCudaErrors(hipMalloc(&d_total_sum, sizeof(unsigned int)));
		checkCudaErrors(hipMemset(d_total_sum, 0, sizeof(unsigned int)));
		//block_sum_reduce<<<1, block_sz, sizeof(unsigned int) * max_elems_per_block>>>(d_total_sum, d_block_sums, grid_sz);
		reduce0<<<1, block_sz, sizeof(unsigned int) * block_sz>>>(d_total_sum, d_block_sums, grid_sz);
		checkCudaErrors(hipMemcpy(&total_sum, d_total_sum, sizeof(unsigned int), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_total_sum));
	}
	else
	{
		unsigned int* d_in_block_sums;
		checkCudaErrors(hipMalloc(&d_in_block_sums, sizeof(unsigned int) * grid_sz));
		checkCudaErrors(hipMemcpy(d_in_block_sums, d_block_sums, sizeof(unsigned int) * grid_sz, hipMemcpyDeviceToDevice));
		total_sum = gpu_sum_reduce(d_in_block_sums, grid_sz);
		checkCudaErrors(hipFree(d_in_block_sums));
	}

	checkCudaErrors(hipFree(d_block_sums));
	return total_sum;
}
